
#include <hip/hip_runtime.h>
extern "C"{
    
    __global__ void writeToSurface(hipSurfaceObject_t target, int width, int height, char r, char g, char b) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (x < width && y < height) {
            uchar4 data = make_uchar4(r, g, b, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), y);
        }
    }

    __global__ void interleaveRGB(hipSurfaceObject_t target, int width, int height,
            unsigned char *R, unsigned char *G, unsigned char *B )
    {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) {       
            unsigned char valR = R[y * width + x]; 
            unsigned char valG = G[y * width + x]; 
            unsigned char valB = B[y * width + x]; 
            uchar4 data = make_uchar4(valR, valG, valB, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), height -1- y);
        }
    }

    __global__ void correlateWithTex(hipTextureObject_t inputTexture, float* filter, unsigned char* out, 
            int inp_rows, int inp_cols, int filter_rows, 
            int filter_cols, int maxDown, int maxRight, int paddedCols
    ) {
        int moveDown = blockDim.x * blockIdx.x + threadIdx.x;
        int moveRight = blockDim.y * blockIdx.y + threadIdx.y;

        if (moveDown >= maxDown) {
            return;
        } 
        if (moveRight >= maxRight) {
            return;
        }  

        float sum = 0;
        for (int filterRow = 0; filterRow < filter_rows; filterRow++) {
            int inputIdx = moveDown * paddedCols + moveRight + filterRow * paddedCols; 

            for (int filterCol = 0; filterCol < filter_cols; filterCol++) {
                //float4 color = tex2D<float4>(inputTexture, 0, 0);
                //float4 color = tex2D(inputTexture, (moveRight + filterCol) * sizeof(uchar4), inp_rows -1- (moveDown + filterRow)));
                //sum += (((float) input[inputIdx + filterCol]))  * filter[filterRow * filter_cols + filterCol];
            }
        }

        //out[moveDown * inp_cols + moveRight] = (unsigned char) (sum);
    }
}