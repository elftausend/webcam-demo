
#include <hip/hip_runtime.h>
extern "C"{
    
    __global__ void writeToSurface(hipSurfaceObject_t target, int width, int height, char r, char g, char b) {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
        if (x < width && y < height) {
            uchar4 data = make_uchar4(r, g, b, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), y);
        }
    }

    __global__ void interleaveRGB(hipSurfaceObject_t target, int width, int height,
            unsigned char *R, unsigned char *G, unsigned char *B )
    {
        unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) {       
            unsigned char valR = R[y * width + x]; 
            unsigned char valG = G[y * width + x]; 
            unsigned char valB = B[y * width + x]; 
            uchar4 data = make_uchar4(valR, valG, valB, 0xff);
            surf2Dwrite(data, target, x * sizeof(uchar4), height -1- y);
        }
    }

    __global__ void correlateWithTex(hipTextureObject_t inputTexture, float* filter, hipSurfaceObject_t out, 
            int inp_rows, int inp_cols, int filter_rows, 
            int filter_cols, int maxDown, int maxRight, int paddedCols
    ) {
        int moveDown = blockDim.x * blockIdx.x + threadIdx.x;
        int moveRight = blockDim.y * blockIdx.y + threadIdx.y;

        if (moveDown >= maxDown) {
            return;
        } 
        if (moveRight >= maxRight) {
            return;
        }  

        float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        for (int filterRow = 0; filterRow < filter_rows; filterRow++) {
            int inputIdx = moveDown * paddedCols + moveRight + filterRow * paddedCols; 

            for (int filterCol = 0; filterCol < filter_cols; filterCol++) {
                float filterVal = filter[filterRow * filter_cols + filterCol];
                // float filterVal = 1.0 / (float) (filter_cols * filter_rows);

                float4 color = tex2D<float4>(inputTexture, (moveRight + filterCol), inp_rows -1- (moveDown + filterRow));
                sum.x += color.x * filterVal;
                sum.y += color.y * filterVal;
                sum.z += color.z * filterVal;
            }
        }

        uchar4 data = make_uchar4((unsigned char) (sum.x * 255.0f), (unsigned char) (sum.y * 255.0f), (unsigned char) (sum.z * 255.0f), 0xff);

        //uchar4 data = make_uchar4(0, 255, 0, 255);

        //printf("R: %d, G: %d, B: %d, A: %d\n", data.x, data.y, data.z, data.w);
        surf2Dwrite(data, out, moveRight * sizeof(uchar4), inp_rows -1- moveDown);

    }

    __constant__ float filterData[64*64];

    __global__ void correlateWithTexShared(hipTextureObject_t inputTexture, hipSurfaceObject_t out, 
            int inp_rows, int inp_cols, int filter_rows, 
            int filter_cols
    ) {

        __shared__ float4 sharedInput[32 + 23][32 + 23];
        // extern __shared__ float4 sharedInput[];
        int moveDown = blockDim.x * blockIdx.x + threadIdx.x;
        int moveRight = blockDim.y * blockIdx.y + threadIdx.y;

        if (moveDown >= inp_rows) {
            return;
        } 
        if (moveRight >= inp_cols) {
            return;
        } 

        if (threadIdx.x < 32 && threadIdx.y < 32) {
            //sharedInput[threadIdx.y * (blockDim.y + filter_cols) + threadIdx.x] = tex2D<float4>(inputTexture, moveRight, moveDown);
            sharedInput[threadIdx.y][threadIdx.x] = tex2D<float4>(inputTexture, moveRight, moveDown);
            
            if (threadIdx.x < filter_rows) {
                //sharedInput[threadIdx.y * (blockDim.y + filter_cols) + threadIdx.x + blockDim.x] = tex2D<float4>(inputTexture, moveRight, moveDown + blockDim.x);
                sharedInput[threadIdx.y][threadIdx.x + blockDim.x] = tex2D<float4>(inputTexture, moveRight, moveDown + blockDim.x);
            }
            if (threadIdx.y < filter_cols) {
                // sharedInput[(threadIdx.y + blockDim.y) * (blockDim.y + filter_cols) + threadIdx.x] = tex2D<float4>(inputTexture, moveRight + blockDim.y, moveDown);
                sharedInput[threadIdx.y + blockDim.y][threadIdx.x] = tex2D<float4>(inputTexture, moveRight + blockDim.y, moveDown);
            }
            if (threadIdx.x < filter_rows && threadIdx.y < filter_cols) {
                // sharedInput[(threadIdx.y + blockDim.y) * (blockDim.y + filter_cols) + threadIdx.x + blockDim.x] = tex2D<float4>(inputTexture, moveRight + blockDim.y, moveDown + blockDim.x);
                sharedInput[threadIdx.y + blockDim.y][threadIdx.x + blockDim.x] = tex2D<float4>(inputTexture, moveRight + blockDim.y, moveDown + blockDim.x);
            }

        }

        __syncthreads();

        float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        for (int filterRow = 0; filterRow < filter_rows; filterRow++) {
            int inputIdx = moveDown * inp_cols + moveRight + filterRow * inp_cols;
 
            for (int filterCol = 0; filterCol < filter_cols; filterCol++) {
                float filterVal = filterData[filterRow * filter_cols + filterCol];
                //float filterVal = 1.0 / (float) (filter_cols * filter_rows);

                //float4 color = sharedInput[(threadIdx.y + filterCol) * (blockDim.y + filter_cols) + threadIdx.x + filterRow];
                float4 color = sharedInput[threadIdx.y + filterCol][threadIdx.x + filterRow];
                sum.x += color.x * filterVal;
                sum.y += color.y * filterVal;
                sum.z += color.z * filterVal;
            }
        }

        uchar4 data = make_uchar4((unsigned char) (sum.x * 255.0f), (unsigned char) (sum.y * 255.0f), (unsigned char) (sum.z * 255.0f), 0xff);

        //uchar4 data = make_uchar4(0, 255, 0, 255);

        //printf("R: %d, G: %d, B: %d, A: %d\n", data.x, data.y, data.z, data.w);
        surf2Dwrite(data, out, moveRight * sizeof(uchar4), moveDown);
    }

    __global__ void correlateShared(unsigned char* input, unsigned char* out, 
            int inp_rows, int inp_cols, int filter_rows, 
            int filter_cols, int maxDown, int maxRight
    ) {
        // make filter size constant?, or use extern shared
        __shared__ unsigned char sharedInput[32 + 16][32 + 16];

        int moveDown = blockDim.x * blockIdx.x + threadIdx.x;
        int moveRight = blockDim.y * blockIdx.y + threadIdx.y;

        if (moveDown >= maxDown) {
            return;
        }
        if (moveRight >= maxRight) {
            return;
        }

        // 32 + filter_rows, 32 + filter_cols ==> 32 + 16, 32 + 16, however, theadIdx max is 32
        if (threadIdx.x < 32 && threadIdx.y < 32) {
            sharedInput[threadIdx.x][threadIdx.y] = input[moveDown * inp_cols + moveRight];
            
            if (threadIdx.x < filter_rows) {
                sharedInput[threadIdx.x + blockDim.x][threadIdx.y] = input[(moveDown + blockDim.x) * inp_cols + moveRight];
            }
            if (threadIdx.y < filter_cols) {
                sharedInput[threadIdx.x][threadIdx.y + blockDim.y] = input[moveDown * inp_cols + moveRight + blockDim.y];
            }
            if (threadIdx.x < filter_rows && threadIdx.y < filter_cols) {
                sharedInput[threadIdx.x + blockDim.x][threadIdx.y + blockDim.y] = input[(moveDown + blockDim.x) * inp_cols + moveRight + blockDim.y];
            }

        }
        __syncthreads();
        float sum = 0;
        for (int filterRow = 0; filterRow < filter_rows; filterRow++) {
            int inputIdx = moveDown * inp_cols + moveRight + filterRow * inp_cols; 
            if (inputIdx >= inp_rows * inp_cols) {
                continue;
            }

            for (int filterCol = 0; filterCol < filter_cols; filterCol++) {
                sum += ((float) sharedInput[threadIdx.x + filterRow][threadIdx.y + filterCol]) * filterData[filterRow * filter_cols + filterCol];                
            }
        }
        // printf("sum: %f\n", sum);
        out[moveDown * inp_cols + moveRight] = (unsigned char) (sum);
    }
}